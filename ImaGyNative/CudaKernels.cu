#include "hip/hip_runtime.h"
#include "CudaKernels.h"
#include <hip/hip_runtime.h>
#include <>

// Helper macro for checking CUDA errors
#define CUDA_CHECK(err_code) do { 
    hipError_t _err = (err_code); 
    if (_err != hipSuccess) { 
        /* In a real app, you\'d want to log this error. */ 
        /* For now, we just return false to signal failure. */ 
        goto error_exit; 
    } 
} while (0)

// ---------------- Pixel-wise Operations ----------------
__global__ void BinarizationKernel(unsigned char* data, int width, int height, int stride, unsigned char threshold) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    int idx = y * stride + x;
    data[idx] = (data[idx] > threshold) ? 255 : 0;
}

bool LaunchBinarizationKernel(unsigned char* pixels, int width, int height, int stride, int threshold)
{
    unsigned char* d_pixels = nullptr;
    size_t imageSize = (size_t)height * stride;

    // 1. Allocate device memory
    CUDA_CHECK(hipMalloc(&d_pixels, imageSize));

    // 2. Copy data from host to device
    CUDA_CHECK(hipMemcpy(d_pixels, pixels, imageSize, hipMemcpyHostToDevice));

    // 3. Set up grid and block dimensions
    dim3 block(16, 16);
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

    // 4. Launch the kernel
    BinarizationKernel<<<grid, block>>>(d_pixels, width, height, stride, (unsigned char)threshold);
    
    // Check for kernel launch errors
    hipError_t lastError = hipGetLastError();
    CUDA_CHECK(lastError);

    // 5. Wait for the kernel to finish
    CUDA_CHECK(hipDeviceSynchronize());

    // 6. Copy data from device back to host
    CUDA_CHECK(hipMemcpy(pixels, d_pixels, imageSize, hipMemcpyDeviceToHost));

    // 7. Free device memory
    // Clean up resources
    if (d_pixels) {
        hipFree(d_pixels);
    }
    return true; // Success

error_exit:
    if (d_pixels) {
        hipFree(d_pixels);
    }
    return false; // Failure
}