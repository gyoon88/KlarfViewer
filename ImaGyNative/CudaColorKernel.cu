#include "hip/hip_runtime.h"
﻿// 파일 이름: CudaColorKernel.cu

#include "CudaColorKernel.cuh"
#include "CudaKernel.cuh" // 흑백용 FFT 커널 등을 재사용하기 위해 포함
#include <hip/hip_runtime.h>
#include <>
#include <vector>
#include <numeric>
#include <algorithm>
#include <cmath>
#include <hipfft/hipfft.h>

namespace ImaGyNative
{
    // --- Helper Macro & Constant Memory ---
#define CUDA_CHECK(err_code) do { hipError_t _err = (err_code); if (_err != hipSuccess) { return false; } } while (0)
#define CUFFT_CHECK(err_code) do { hipfftResult _err = (err_code); if (_err != HIPFFT_SUCCESS) { return false; } } while (0)

// 가우시안, 평균 필터 커널을 저장하기 위한 상수 메모리
    __constant__ float c_colorFilterKernel[625];

    // CPU에서 float 타입의 커널을 생성하는 헬퍼 함수 (흑백용 코드에서 가져옴)
    std::vector<float> createGaussianKernelFloat(int kernelSize, double sigma, bool isCircular);
    std::vector<float> createAverageKernelFloat(int kernelSize, bool isCircular);

    // ==========================================
    // --- CUDA 컬러 커널 정의 ---
    // ==========================================

    // 컬러 컨볼루션 커널 (가우시안, 평균 필터)
    __global__ void ColorConvolutionKernel(const uchar4* input, uchar4* output, int width, int height, int kernelSize)
    {
        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x >= width || y >= height) return;

        int center = kernelSize / 2;
        float sumB = 0.0f, sumG = 0.0f, sumR = 0.0f;

        for (int ky = -center; ky <= center; ++ky) {
            for (int kx = -center; kx <= center; ++kx) {
                int nX = x + kx;
                int nY = y + ky;

                if (nX >= 0 && nX < width && nY >= 0 && nY < height) {
                    float weight = c_colorFilterKernel[(ky + center) * kernelSize + (kx + center)];
                    uchar4 pixel = input[nY * width + nX];
                    sumB += pixel.x * weight; // .x는 B
                    sumG += pixel.y * weight; // .y는 G
                    sumR += pixel.z * weight; // .z는 R
                }
            }
        }

        uchar4 outPixel;
        outPixel.x = (unsigned char)fmaxf(0.f, fminf(255.f, sumB));
        outPixel.y = (unsigned char)fmaxf(0.f, fminf(255.f, sumG));
        outPixel.z = (unsigned char)fmaxf(0.f, fminf(255.f, sumR));
        outPixel.w = input[y * width + x].w; // Alpha 채널은 보존

        output[y * width + x] = outPixel;
    }

    // 컬러 팽창 커널
    __global__ void ColorDilationKernel(const uchar4* input, uchar4* output, int width, int height, int kernelSize, bool useCircularKernel) // 파라미터 추가
    {
        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x >= width || y >= height) return;

        int center = kernelSize / 2;
        unsigned char maxB = 0, maxG = 0, maxR = 0;

        for (int ky = -center; ky <= center; ++ky) {
            for (int kx = -center; kx <= center; ++kx) {
                // ✨ 추가된 부분: 원형 커널일 경우, 원 밖의 픽셀은 건너뜁니다.
                if (useCircularKernel && (kx * kx + ky * ky) > (center * center)) {
                    continue;
                }

                int nX = x + kx;
                int nY = y + ky;
                if (nX >= 0 && nX < width && nY >= 0 && nY < height) {
                    uchar4 pixel = input[nY * width + nX];
                    maxB = max(maxB, pixel.x);
                    maxG = max(maxG, pixel.y);
                    maxR = max(maxR, pixel.z);
                }
            }
        }
        output[y * width + x] = make_uchar4(maxB, maxG, maxR, input[y * width + x].w);
    }

    // 컬러 침식 커널
    __global__ void ColorErosionKernel(const uchar4* input, uchar4* output, int width, int height, int kernelSize, bool useCircularKernel) 
    {
        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y = blockIdx.y * blockDim.y + threadIdx.y;

        if (x >= width || y >= height) return;

        int center = kernelSize / 2;
        unsigned char minB = 255, minG = 255, minR = 255;

        for (int ky = -center; ky <= center; ++ky) {
            for (int kx = -center; kx <= center; ++kx) {
                // 원형 커널일 경우, 원 밖의 픽셀은 건너뜀
                if (useCircularKernel && (kx * kx + ky * ky) > (center * center)) {
                    continue;
                }

                int nX = x + kx;
                int nY = y + ky;
                if (nX >= 0 && nX < width && nY >= 0 && nY < height) {
                    uchar4 pixel = input[nY * width + nX];
                    minB = min(minB, pixel.x);
                    minG = min(minG, pixel.y);
                    minR = min(minR, pixel.z);
                }
            }
        }
        output[y * width + x] = make_uchar4(minB, minG, minR, input[y * width + x].w);
    }

    // BGRA -> 3채널(B,G,R)로 분리하는 커널
    __global__ void SplitBGRAKernel(const uchar4* input, unsigned char* b_out, unsigned char* g_out, unsigned char* r_out, int N) {
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i < N) {
            uchar4 pixel = input[i];
            b_out[i] = pixel.x;
            g_out[i] = pixel.y;
            r_out[i] = pixel.z;
        }
    }

    // 3채널(B,G,R) -> BGRA로 병합하는 커널
    __global__ void MergeToBGRAKernel(unsigned char* bgra_out, const unsigned char* b_in, const unsigned char* g_in, const unsigned char* r_in, int N) {
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i < N) {
            // uchar4*로 캐스팅하여 4바이트 단위로 쓰기
            ((uchar4*)bgra_out)[i] = make_uchar4(b_in[i], g_in[i], r_in[i], 255);
        }
    }


    // ==========================================
    // --- CUDA 컬러 Launcher 함수 구현 ---
    // ==========================================

    bool LaunchGaussianBlurColorKernel(unsigned char* pixels, int width, int height, int stride, double sigma, int kernelSize, bool useCircularKernel) {
        if (stride != width * 4) return false;

        size_t imageSize = (size_t)height * stride;
        uchar4* d_input = nullptr, * d_output = nullptr;
        std::vector<float> h_kernel = createGaussianKernelFloat(kernelSize, sigma, useCircularKernel);

        CUDA_CHECK(hipMalloc(&d_input, imageSize));
        CUDA_CHECK(hipMalloc(&d_output, imageSize));
        CUDA_CHECK(hipMemcpy(d_input, pixels, imageSize, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_colorFilterKernel), h_kernel.data(), h_kernel.size() * sizeof(float)));

        dim3 block(16, 16);
        dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

        ColorConvolutionKernel << <grid, block >> > ((uchar4*)d_input, (uchar4*)d_output, width, height, kernelSize);

        CUDA_CHECK(hipDeviceSynchronize());
        CUDA_CHECK(hipMemcpy(pixels, d_output, imageSize, hipMemcpyDeviceToHost));

        hipFree(d_input);
        hipFree(d_output);
        return true;
    }

    bool LaunchAverageBlurColorKernel(unsigned char* pixels, int width, int height, int stride, int kernelSize, bool useCircularKernel) {
        // LaunchGaussianBlurColorKernel과 거의 동일하고 h_kernel 생성 부분만 다름
        if (stride != width * 4) return false;

        size_t imageSize = (size_t)height * stride;
        uchar4* d_input = nullptr, * d_output = nullptr;
        std::vector<float> h_kernel = createAverageKernelFloat(kernelSize, useCircularKernel);

        CUDA_CHECK(hipMalloc(&d_input, imageSize));
        CUDA_CHECK(hipMalloc(&d_output, imageSize));
        CUDA_CHECK(hipMemcpy(d_input, pixels, imageSize, hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_colorFilterKernel), h_kernel.data(), h_kernel.size() * sizeof(float)));

        dim3 block(16, 16);
        dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

        ColorConvolutionKernel << <grid, block >> > ((uchar4*)d_input, (uchar4*)d_output, width, height, kernelSize);

        CUDA_CHECK(hipDeviceSynchronize());
        CUDA_CHECK(hipMemcpy(pixels, d_output, imageSize, hipMemcpyDeviceToHost));

        hipFree(d_input);
        hipFree(d_output);
        return true;
    }

    bool LaunchDilationColorKernel(unsigned char* pixels, int width, int height, int stride, int kernelSize, bool useCircularKernel) {
        if (stride != width * 4) return false;
        size_t imageSize = (size_t)height * stride;
        uchar4* d_input = nullptr, * d_output = nullptr;

        CUDA_CHECK(hipMalloc(&d_input, imageSize));
        CUDA_CHECK(hipMalloc(&d_output, imageSize));
        CUDA_CHECK(hipMemcpy(d_input, pixels, imageSize, hipMemcpyHostToDevice));

        dim3 block(16, 16);
        dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

        ColorDilationKernel << <grid, block >> > ((uchar4*)d_input, (uchar4*)d_output, width, height, kernelSize, useCircularKernel);

        CUDA_CHECK(hipDeviceSynchronize());
        CUDA_CHECK(hipMemcpy(pixels, d_output, imageSize, hipMemcpyDeviceToHost));

        hipFree(d_input);
        hipFree(d_output);
        return true;
    }

    bool LaunchErosionColorKernel(unsigned char* pixels, int width, int height, int stride, int kernelSize, bool useCircularKernel) {
        if (stride != width * 4) return false;
        size_t imageSize = (size_t)height * stride;
        uchar4* d_input = nullptr, * d_output = nullptr;

        CUDA_CHECK(hipMalloc(&d_input, imageSize));
        CUDA_CHECK(hipMalloc(&d_output, imageSize));
        CUDA_CHECK(hipMemcpy(d_input, pixels, imageSize, hipMemcpyHostToDevice));

        dim3 block(16, 16);
        dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

        ColorErosionKernel << <grid, block >> > ((uchar4*)d_input, (uchar4*)d_output, width, height, kernelSize, useCircularKernel);

        CUDA_CHECK(hipDeviceSynchronize());
        CUDA_CHECK(hipMemcpy(pixels, d_output, imageSize, hipMemcpyDeviceToHost));

        hipFree(d_input);
        hipFree(d_output);
        return true;
    }

    bool LaunchEqualizationColorKernel(unsigned char* pixels, int width, int height, int stride) {
        if (stride != width * 4) return false;
        const int N = width * height;
        size_t imageSize = (size_t)height * stride;
        size_t channelSize = N * sizeof(unsigned char);

        uchar4* d_input = nullptr;
        unsigned char* d_b = nullptr, * d_g = nullptr, * d_r = nullptr;

        CUDA_CHECK(hipMalloc(&d_input, imageSize));
        CUDA_CHECK(hipMalloc(&d_b, channelSize));
        CUDA_CHECK(hipMalloc(&d_g, channelSize));
        CUDA_CHECK(hipMalloc(&d_r, channelSize));
        CUDA_CHECK(hipMemcpy(d_input, pixels, imageSize, hipMemcpyHostToDevice));

        // BGRA -> B, G, R 세 채널로 분리
        dim3 grid1D((N + 255) / 256);
        dim3 block1D(256);
        SplitBGRAKernel << <grid1D, block1D >> > ((uchar4*)d_input, d_b, d_g, d_r, N);

        // 각 채널에 흑백용 평활화 함수 재사용
        LaunchEqualizationKernel(d_b, width, height, width);
        LaunchEqualizationKernel(d_g, width, height, width);
        LaunchEqualizationKernel(d_r, width, height, width);

        // 처리된 B, G, R 채널을 다시 BGRA로 병합
        MergeToBGRAKernel << <grid1D, block1D >> > ((unsigned char*)d_input, d_b, d_g, d_r, N);

        CUDA_CHECK(hipDeviceSynchronize());
        CUDA_CHECK(hipMemcpy(pixels, d_input, imageSize, hipMemcpyDeviceToHost));

        hipFree(d_input);
        hipFree(d_b);
        hipFree(d_g);
        hipFree(d_r);
        return true;
    }

    bool LaunchFftSpectrumColorKernel(unsigned char* pixels, int width, int height, int stride) {
        if (stride != width * 4) return false;
        const int N = width * height;
        size_t imageSize = (size_t)height * stride;
        size_t channelSize = N * sizeof(unsigned char);

        uchar4* d_input = nullptr;
        unsigned char* d_b = nullptr, * d_g = nullptr, * d_r = nullptr;

        CUDA_CHECK(hipMalloc(&d_input, imageSize));
        CUDA_CHECK(hipMalloc(&d_b, channelSize));
        CUDA_CHECK(hipMalloc(&d_g, channelSize));
        CUDA_CHECK(hipMalloc(&d_r, channelSize));
        CUDA_CHECK(hipMemcpy(d_input, pixels, imageSize, hipMemcpyHostToDevice));

        // 1. BGRA -> B, G, R 채널 분리
        dim3 grid1D((N + 255) / 256);
        dim3 block1D(256);
        SplitBGRAKernel << <grid1D, block1D >> > ((uchar4*)d_input, d_b, d_g, d_r, N);

        // 각 채널에 흑백용 FFT 스펙트럼 함수 재사용
        LaunchFftSpectrumKernel(d_b, width, height, width);
        LaunchFftSpectrumKernel(d_g, width, height, width);
        LaunchFftSpectrumKernel(d_r, width, height, width);

        // 처리된 B, G, R 채널을 다시 BGRA로 병합
        MergeToBGRAKernel << <grid1D, block1D >> > ((unsigned char*)d_input, d_b, d_g, d_r, N);

        CUDA_CHECK(hipDeviceSynchronize());
        CUDA_CHECK(hipMemcpy(pixels, d_input, imageSize, hipMemcpyDeviceToHost));

        hipFree(d_input);
        hipFree(d_b);
        hipFree(d_g);
        hipFree(d_r);
        return true;
    }
}