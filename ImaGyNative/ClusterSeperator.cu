
#include <hip/hip_runtime.h>
//#include "pch.h"
//#include "CudaKernel.cuh"
//#include <cuda_runtime.h>
//#include <device_launch_parameters.h>
//#include <vector>
//#include <numeric>
//#include <algorithm>
//#include <cmath>
//#include <cufft.h> // For using cuFFT  
//#include <random>
//#include <limits>
//
//namespace ImaGyNative {
//
//	//==========================================
//	//-- - K - Means Clustering Ŀ�� ����-- -
//	//==========================================
//	struct Point5D {
//		float r, g, b, x, y;
//	};
//#define CUDA_CHECK(err_code) do { cudaError_t _err = (err_code); if (_err != cudaSuccess) { return false; } } while (0)
//	// --- NEW: cuFFT vailable state macro
//#define CUFFT_CHECK(err_code) do { cufftResult _err = (err_code); if (_err != CUFFT_SUCCESS) { return false; } } while (0)
//
////�Ҵ� �ܰ� : �� �ȼ��� ���� ����� ��ǥ���� �Ҵ��ϴ� Ŀ��
//	__global__ void KMeansAssignmentKernel(const Point5D* normalizedPixels, const Point5D* centroids, int* assignments, int numPixels, int k) {
//		int i = blockIdx.x * blockDim.x + threadIdx.x;
//		if (i >= numPixels) return;
//
//		float minDistSq = 1e10f; // ����� ū ������ �ʱ�ȭ
//		int bestCluster = 0;
//
//		for (int c = 0; c < k; ++c) {
//			float dr = normalizedPixels[i].r - centroids[c].r;
//			float dg = normalizedPixels[i].g - centroids[c].g;
//			float db = normalizedPixels[i].b - centroids[c].b;
//			float dx = normalizedPixels[i].x - centroids[c].x;
//			float dy = normalizedPixels[i].y - centroids[c].y;
//			float distSq = dr * dr + dg * dg + db * db + dx * dx + dy * dy;
//
//			if (distSq < minDistSq) {
//				minDistSq = distSq;
//				bestCluster = c;
//			}
//		}
//		assignments[i] = bestCluster;
//	}
//	//������Ʈ �ܰ� : �� Ŭ�������� �հ�� ī��Ʈ�� ����ϴ� Ŀ��
//	__global__ void KMeansUpdateKernel(const Point5D* normalizedPixels, const int* assignments,
//		Point5D* newCentroids, int* counts, int numPixels) {
//		int i = blockIdx.x * blockDim.x + threadIdx.x;
//		if (i >= numPixels) return;
//
//		int clusterId = assignments[i];
//
//		// atomicAdd�� ����Ͽ� ���� �����尡 ���ÿ� �����ϰ� ���� ���� �� �ֵ��� ��
//		atomicAdd(&newCentroids[clusterId].r, normalizedPixels[i].r);
//		atomicAdd(&newCentroids[clusterId].g, normalizedPixels[i].g);
//		atomicAdd(&newCentroids[clusterId].b, normalizedPixels[i].b);
//		atomicAdd(&newCentroids[clusterId].x, normalizedPixels[i].x);
//		atomicAdd(&newCentroids[clusterId].y, normalizedPixels[i].y);
//		atomicAdd(&counts[clusterId], 1);
//	}
//
//	// ���� �ܰ� : �ջ�� ���� �������� ���ο� ��ǥ���� ����� ����ϴ� Ŀ��
//	__global__ void KMeansFinalizeCentroidsKernel(Point5D* centroids, const Point5D* newCentroids,
//		const int* counts, int k) {
//		int c = blockIdx.x * blockDim.x + threadIdx.x;
//		if (c >= k) return;
//
//		if (counts[c] > 0) {
//			centroids[c].r = newCentroids[c].r / counts[c];
//			centroids[c].g = newCentroids[c].g / counts[c];
//			centroids[c].b = newCentroids[c].b / counts[c];
//			centroids[c].x = newCentroids[c].x / counts[c];
//			centroids[c].y = newCentroids[c].y / counts[c];
//		}
//	}
//
//	// ���� ������ : �� �ȼ��� ���� ��ǥ �������� ĥ�ϴ� Ŀ��
//	__global__ void KMeansRenderKernel(unsigned char* pixels, const int* assignments, const Point5D* centroids, int width, int height, int stride) {
//		int x = blockIdx.x * blockDim.x + threadIdx.x;
//		int y = blockIdx.y * blockDim.y + threadIdx.y;
//
//		if (x >= width || y >= height) return;
//
//		int clusterId = assignments[y * width + x];
//		unsigned char* p = pixels + y * stride + x * 4;
//
//		p[2] = (unsigned char)(centroids[clusterId].r * 255.0f); // R
//		p[1] = (unsigned char)(centroids[clusterId].g * 255.0f); // G
//		p[0] = (unsigned char)(centroids[clusterId].b * 255.0f); // B
//	}
//
//	// --- K-Means CUDA Launcher �Լ� ---
//	bool LaunchKMeansKernel(void* pixels, int width, int height, int stride, int k, int iteration) {
//		unsigned char* h_pixels = static_cast<unsigned char*>(pixels);
//		int numPixels = width * height;
//
//		// GPU �޸� �Ҵ�
//		unsigned char* d_pixels;
//		Point5D* d_normalizedPixels;
//		Point5D* d_centroids;
//		int* d_assignments;
//		Point5D* d_newCentroids;
//		int* d_counts;
//
//		CUDA_CHECK(cudaMalloc(&d_pixels, (size_t)height * stride));
//		CUDA_CHECK(cudaMalloc(&d_normalizedPixels, (size_t)numPixels * sizeof(Point5D)));
//		CUDA_CHECK(cudaMalloc(&d_centroids, (size_t)k * sizeof(Point5D)));
//		CUDA_CHECK(cudaMalloc(&d_assignments, (size_t)numPixels * sizeof(int)));
//		CUDA_CHECK(cudaMalloc(&d_newCentroids, (size_t)k * sizeof(Point5D)));
//		CUDA_CHECK(cudaMalloc(&d_counts, (size_t)k * sizeof(int)));
//
//
//		// ������ �غ�: CPU���� ����ȭ �� GPU�� ����
//		std::vector<Point5D> h_normalizedPixels(numPixels);
//		double w_minus_1 = width > 1 ? (double)(width - 1) : 1.0;
//		double h_minus_1 = height > 1 ? (double)(height - 1) : 1.0;
//
//#pragma omp parallel for
//		for (int y = 0; y < height; ++y) {
//			for (int x = 0; x < width; ++x) {
//				unsigned char* p = h_pixels + y * stride + x * 4;
//				h_normalizedPixels[y * width + x] = {
//					(float)(p[2] / 255.0), (float)(p[1] / 255.0), (float)(p[0] / 255.0),
//					(float)(x / w_minus_1), (float)(y / h_minus_1)
//				};
//			}
//		}
//		CUDA_CHECK(cudaMemcpy(d_normalizedPixels, h_normalizedPixels.data(), (size_t)numPixels * sizeof(Point5D), cudaMemcpyHostToDevice));
//
//		// �ʱ� ��ǥ�� ���� �� GPU�� ����
//		std::vector<Point5D> h_centroids(k);
//		std::mt19937 rng(std::random_device{}());
//		std::uniform_int_distribution<int> dist(0, numPixels - 1);
//		for (int i = 0; i < k; ++i) {
//			h_centroids[i] = h_normalizedPixels[dist(rng)];
//		}
//		CUDA_CHECK(cudaMemcpy(d_centroids, h_centroids.data(), (size_t)k * sizeof(Point5D), cudaMemcpyHostToDevice));
//
//		// �׸��� �� ��� ����
//
//		dim3 block1D_pixels(256);
//		dim3 grid1D_pixels((numPixels + block1D_pixels.x - 1) / block1D_pixels.x);
//
//		dim3 block1D_k(256);
//		dim3 grid1D_k((k + block1D_k.x - 1) / block1D_k.x);
//
//		// K-Means �ݺ� (������ �ٽ� ����)
//		for (int i = 0; i < iteration; ++i) {
//			// 1. �Ҵ� �ܰ� Ŀ�� ���� (������ ����)
//			KMeansAssignmentKernel << <grid1D_pixels, block1D_pixels >> > (d_normalizedPixels, d_centroids, d_assignments, numPixels, k);
//			CUDA_CHECK(cudaGetLastError());
//
//			// GPU �޸� �ʱ�ȭ
//			CUDA_CHECK(cudaMemset(d_newCentroids, 0, (size_t)k * sizeof(Point5D)));
//			CUDA_CHECK(cudaMemset(d_counts, 0, (size_t)k * sizeof(int)));
//
//			// 2. ������Ʈ �ܰ� Ŀ�� ���� (CPU ��� ��ü)
//			KMeansUpdateKernel << <grid1D_pixels, block1D_pixels >> > (d_normalizedPixels, d_assignments, d_newCentroids, d_counts, numPixels);
//			CUDA_CHECK(cudaGetLastError());
//
//			// 3. ���� ��ǥ�� ��� Ŀ�� ����
//			KMeansFinalizeCentroidsKernel << <grid1D_k, block1D_k >> > (d_centroids, d_newCentroids, d_counts, k);
//			CUDA_CHECK(cudaGetLastError());
//		}
//
//		// ���� ������
//		CUDA_CHECK(cudaMemcpy(d_pixels, h_pixels, (size_t)height * stride, cudaMemcpyHostToDevice)); // ���� �̹��� ������ ����
//		dim3 grid2D((width + 15) / 16, (height + 15) / 16);
//		dim3 block2D(16, 16);
//		KMeansRenderKernel << <grid2D, block2D >> > (d_pixels, d_assignments, d_centroids, width, height, stride);
//		CUDA_CHECK(cudaGetLastError());
//		CUDA_CHECK(cudaDeviceSynchronize());
//
//		// ���� ����� ȣ��Ʈ �޸𸮷� ����
//		CUDA_CHECK(cudaMemcpy(pixels, d_pixels, (size_t)height * stride, cudaMemcpyDeviceToHost));
//
//		// GPU �޸� ����
//		cudaFree(d_pixels);
//		cudaFree(d_normalizedPixels);
//		cudaFree(d_centroids);
//		cudaFree(d_assignments);
//		cudaFree(d_newCentroids);
//		cudaFree(d_counts);
//
//		return true;
//	}
//}