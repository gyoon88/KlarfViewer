#include "hip/hip_runtime.h"
#include "CudaKernel.cuh"
#include <hip/hip_runtime.h>
#include <>
#include <vector>
#include <numeric>
#include <algorithm>
#include <cmath>
#include <hipfft/hipfft.h> // For using cuFFT  
#include <random>
#include <limits>

namespace ImaGyNative
{
	// --- Helper Macro & Constant Memory (수정된 버전) ---
#define CUDA_CHECK(err_code) do { \
    hipError_t _err = (err_code); \
    if (_err != hipSuccess) { \
        printf("CUDA Error at %s:%d - %s\n", __FILE__, __LINE__, hipGetErrorString(_err)); \
        return false; \
    } \
} while (0)

#define CUFFT_CHECK(err_code) do { \
    hipfftResult _err = (err_code); \
    if (_err != HIPFFT_SUCCESS) { \
        /* cuFFT는 에러 문자열을 바로 반환하는 함수가 없으므로 에러 코드를 직접 출력합니다. */ \
        printf("cuFFT Error at %s:%d - error code %d\n", __FILE__, __LINE__, _err); \
        return false; \
    } \
} while (0)
//// --- Helper Macro & Constant Memory ---
//#define CUDA_CHECK(err_code) do { hipError_t _err = (err_code); if (_err != hipSuccess) { return false; } } while (0)
//// --- NEW: cuFFT vailable state macro
//#define CUFFT_CHECK(err_code) do { hipfftResult _err = (err_code); if (_err != HIPFFT_SUCCESS) { return false; } } while (0)

	__constant__ float c_filterKernel[625];
	__constant__ float c_sobelKernelX[625];
	__constant__ float c_sobelKernelY[625];

	// Convolution Kernel Generating Method by CPU!!
	std::vector<float> createGaussianKernelFloat(int kernelSize, double sigma, bool isCircular) {
		if (kernelSize % 2 == 0) kernelSize++;
		std::vector<float> kernel(kernelSize * kernelSize);
		float sum = 0.0f;
		int center = kernelSize / 2;
		double radiusSq = center * center;
		const float M_PI_F = 3.1415926535f;
		for (int y = 0; y < kernelSize; ++y) {
			for (int x = 0; x < kernelSize; ++x) {
				if (isCircular && ((x - center) * (x - center) + (y - center) * (y - center)) > radiusSq) {
					kernel[y * kernelSize + x] = 0.0f;
					continue;
				}
				int dx = x - center;
				int dy = y - center;
				float val = expf(-(dx * dx + dy * dy) / (2.0f * (float)sigma * (float)sigma)) / (2.0f * M_PI_F * (float)sigma * (float)sigma);
				kernel[y * kernelSize + x] = val;
				sum += val;
			}
		}
		if (sum > 0) { for (float& val : kernel) { val /= sum; } }
		return kernel;
	}
	std::vector<float> createAverageKernelFloat(int kernelSize, bool isCircular) {
		if (kernelSize % 2 == 0) kernelSize++;
		std::vector<float> kernel(kernelSize * kernelSize, 0.0f);
		int center = kernelSize / 2;
		double radiusSq = center * center;
		float count = 0.0f;
		for (int y = 0; y < kernelSize; ++y) {
			for (int x = 0; x < kernelSize; ++x) {
				if (isCircular) {
					if (((x - center) * (x - center) + (y - center) * (y - center)) <= radiusSq) {
						kernel[y * kernelSize + x] = 1.0f;
						count++;
					}
				}
				else {
					kernel[y * kernelSize + x] = 1.0f;
					count++;
				}
			}
		}
		if (count > 0) { for (float& val : kernel) { val /= count; } }
		return kernel;
	}
	void createSobelKernelsFloat(std::vector<float>& kernelX, std::vector<float>& kernelY, int kernelSize) {
		if (kernelSize % 2 == 0) kernelSize++;
		kernelX.assign(kernelSize * kernelSize, 0.0f);
		kernelY.assign(kernelSize * kernelSize, 0.0f);
		int center = kernelSize / 2;
		for (int y = 0; y < kernelSize; ++y) {
			for (int x = 0; x < kernelSize; ++x) {
				int dx = x - center;
				int dy = y - center;
				float denom = (float)(dx * dx + dy * dy);
				if (denom > 0) {
					if (dx != 0) kernelX[y * kernelSize + x] = dx / denom;
					if (dy != 0) kernelY[y * kernelSize + x] = dy / denom;
				}
			}
		}
	}
	std::vector<float> createLaplacianKernelFloat(int kernelSize) {
		if (kernelSize % 2 == 0) kernelSize++;
		std::vector<float> kernel(kernelSize * kernelSize, -1.0f);
		int centerIndex = (kernelSize / 2) * kernelSize + (kernelSize / 2);
		kernel[centerIndex] = (float)(kernelSize * kernelSize - 1);
		return kernel;
	}

	// ==========================================
	// CUDA Parallel Kernels
	// ==========================================
#define TILE_DIM 16

	// Optimize memory velocity for the Process that uses convolution kernel 
	__device__ void loadTile(const unsigned char* input, unsigned char* tile, int width, int height, int stride, int kernelSize) {
		int center = kernelSize / 2;
		int tx = threadIdx.x;
		int ty = threadIdx.y;

		for (int i = ty; i < TILE_DIM + 2 * center; i += TILE_DIM) {
			for (int j = tx; j < TILE_DIM + 2 * center; j += TILE_DIM) {
				int loadX = blockIdx.x * TILE_DIM - center + j;
				int loadY = blockIdx.y * TILE_DIM - center + i;
				if (loadX >= 0 && loadX < width && loadY >= 0 && loadY < height) {
					tile[i * (TILE_DIM + 14) + j] = input[loadY * stride + loadX];
				}
				else {
					tile[i * (TILE_DIM + 14) + j] = 0; // Padding
				}
			}
		}
	}

	// ==========================================
	// --- Binarization 커널 정의 ---
	// ==========================================
	__global__ void BinarizationKernel(unsigned char* data, int width, int height, int stride, unsigned char threshold) {
		int x = blockIdx.x * blockDim.x + threadIdx.x;
		int y = blockIdx.y * blockDim.y + threadIdx.y;
		if (x >= width || y >= height) return;
		int idx = y * stride + x;
		data[idx] = (data[idx] > threshold) ? 255 : 0;
	}


	// ==========================================
	// --- Convolustion computing 커널 정의 ---
	// ==========================================
	__global__ void ConvolutionSharedMemKernel(const unsigned char* input, unsigned char* output, int width, int height, int stride, int kernelSize) {
		__shared__ unsigned char tile[(TILE_DIM + 14) * (TILE_DIM + 14)];
		loadTile(input, tile, width, height, stride, kernelSize);
		__syncthreads();

		int outX = blockIdx.x * TILE_DIM + threadIdx.x;
		int outY = blockIdx.y * TILE_DIM + threadIdx.y;

		if (outX < width && outY < height) {
			float sum = 0.0f;
			for (int ky = 0; ky < kernelSize; ++ky) {
				for (int kx = 0; kx < kernelSize; ++kx) {
					sum += c_filterKernel[ky * kernelSize + kx] * tile[(threadIdx.y + ky) * (TILE_DIM + 14) + (threadIdx.x + kx)];
				}
			}
			output[outY * stride + outX] = (unsigned char)fmaxf(0.f, fminf(255.f, sum));
		}
	}
	__global__ void SobelSharedMemKernel(const unsigned char* input, unsigned char* output, int width, int height, int stride, int kernelSize) {
		__shared__ unsigned char tile[(TILE_DIM + 14) * (TILE_DIM + 14)];
		loadTile(input, tile, width, height, stride, kernelSize);
		__syncthreads();

		int outX = blockIdx.x * TILE_DIM + threadIdx.x;
		int outY = blockIdx.y * TILE_DIM + threadIdx.y;

		if (outX < width && outY < height) {
			float sumX = 0.0f, sumY = 0.0f;
			for (int ky = 0; ky < kernelSize; ++ky) {
				for (int kx = 0; kx < kernelSize; ++kx) {
					int kIdx = ky * kernelSize + kx;
					unsigned char pixel_val = tile[(threadIdx.y + ky) * (TILE_DIM + 14) + (threadIdx.x + kx)];
					sumX += c_sobelKernelX[kIdx] * pixel_val;
					sumY += c_sobelKernelY[kIdx] * pixel_val;
				}
			}
			float finalVal = sqrtf(sumX * sumX + sumY * sumY);
			output[outY * stride + outX] = (unsigned char)fmaxf(0.f, fminf(255.f, finalVal));
		}
	}
	__global__ void DilationSharedMemKernel(const unsigned char* input, unsigned char* output, int width, int height, int stride, int kernelSize, bool useCircularKernel) {
		__shared__ unsigned char tile[(TILE_DIM + 14) * (TILE_DIM + 14)];
		loadTile(input, tile, width, height, stride, kernelSize);
		__syncthreads();

		int outX = blockIdx.x * TILE_DIM + threadIdx.x;
		int outY = blockIdx.y * TILE_DIM + threadIdx.y;
		int center = kernelSize / 2;

		if (outX < width && outY < height) {
			unsigned char maxVal = 0;
			for (int ky = 0; ky < kernelSize; ++ky) {
				for (int kx = 0; kx < kernelSize; ++kx) {
					if (useCircularKernel && ((kx - center) * (kx - center) + (ky - center) * (ky - center) > center * center)) continue;
					unsigned char val = tile[(threadIdx.y + ky) * (TILE_DIM + 14) + (threadIdx.x + kx)];
					if (val > maxVal) maxVal = val;
				}
			}
			output[outY * stride + outX] = maxVal;
		}
	}
	__global__ void ErosionSharedMemKernel(const unsigned char* input, unsigned char* output, int width, int height, int stride, int kernelSize, bool useCircularKernel) {
		__shared__ unsigned char tile[(TILE_DIM + 14) * (TILE_DIM + 14)];
		loadTile(input, tile, width, height, stride, kernelSize);
		__syncthreads();

		int outX = blockIdx.x * TILE_DIM + threadIdx.x;
		int outY = blockIdx.y * TILE_DIM + threadIdx.y;
		int center = kernelSize / 2;

		if (outX < width && outY < height) {
			unsigned char minVal = 255;
			for (int ky = 0; ky < kernelSize; ++ky) {
				for (int kx = 0; kx < kernelSize; ++kx) {
					if (useCircularKernel && ((kx - center) * (kx - center) + (ky - center) * (ky - center) > center * center)) continue;
					unsigned char val = tile[(threadIdx.y + ky) * (TILE_DIM + 14) + (threadIdx.x + kx)];
					if (val < minVal) minVal = val;
				}
			}
			output[outY * stride + outX] = minVal;
		}
	}


	// ==========================================
	// --- Equalization 커널 정의 ---
	// ==========================================
	__global__ void Histogram256Kernel(const unsigned char* data, int width, int height, int stride, unsigned int* hist) {
		int x = blockIdx.x * blockDim.x + threadIdx.x;
		int y = blockIdx.y * blockDim.y + threadIdx.y;
		if (x < width && y < height) {
			atomicAdd(&hist[data[y * stride + x]], 1);
		}
	}

	__global__ void EqualizationKernel(unsigned char* data, int width, int height, int stride, const int* cdf, int cdfMin, int totalPixels) {
		int x = blockIdx.x * blockDim.x + threadIdx.x;
		int y = blockIdx.y * blockDim.y + threadIdx.y;
		if (x >= width || y >= height) return;
		int idx = y * stride + x;
		int val = data[idx];
		int newVal = roundf(((float)cdf[val] - cdfMin) / (totalPixels - cdfMin) * 255.0f);
		data[idx] = (unsigned char)fmaxf(0.f, fminf(255.f, (float)newVal));
	}


	// ==========================================
	// --- SAD / SSD CUDA 커널 정의 ---
	// ==========================================

	__global__ void SadKernel(const unsigned char* image, const unsigned char* templ,
		unsigned int* result, 
		int width, int height, int stride,
		int tempWidth, int tempHeight, int tempStride)
	{
		int x = blockIdx.x * blockDim.x + threadIdx.x;
		int y = blockIdx.y * blockDim.y + threadIdx.y;

		if (x > width - tempWidth || y > height - tempHeight) return;

		unsigned int current_sad = 0;
		for (int ty = 0; ty < tempHeight; ++ty) {
			for (int tx = 0; tx < tempWidth; ++tx) {
				unsigned char imagePixel = image[(y + ty) * stride + (x + tx)];
				unsigned char templatePixel = templ[ty * tempStride + tx];
				current_sad += abs(imagePixel - templatePixel);
			}
		}
		// 결과 버퍼에 SAD 값 저장
		result[y * width + x] = current_sad;
	}

	__global__ void SsdKernel(const unsigned char* image, const unsigned char* templ,
		unsigned int* result,
		int width, int height, int stride,
		int tempWidth, int tempHeight, int tempStride)
	{
		int x = blockIdx.x * blockDim.x + threadIdx.x;
		int y = blockIdx.y * blockDim.y + threadIdx.y;

		if (x > width - tempWidth || y > height - tempHeight) return;

		unsigned int current_ssd = 0;

		for (int ty = 0; ty < tempHeight; ++ty) {
			for (int tx = 0; tx < tempWidth; ++tx) {
				int diff = (int)image[(y + ty) * stride + (x + tx)] - (int)templ[ty * tempStride + tx];
				current_ssd += diff * diff;
			}
		}
		result[y * width + x] = current_ssd;
	}

	__global__ void NccKernel(const unsigned char* image, const unsigned char* templ, float* result,
		int width, int height, int stride,
		int tempWidth, int tempHeight, int tempStride,
		double meanT, double stdDevT_inv) {
		int x = blockIdx.x * blockDim.x + threadIdx.x;
		int y = blockIdx.y * blockDim.y + threadIdx.y;

		if (x > width - tempWidth || y > height - tempHeight) return;

		double sumI = 0.0, sumI2 = 0.0, sumIT = 0.0;
		int N = tempWidth * tempHeight;

		for (int ty = 0; ty < tempHeight; ++ty) {
			for (int tx = 0; tx < tempWidth; ++tx) {
				double pixelI = image[(y + ty) * stride + (x + tx)];
				double pixelT = templ[ty * tempStride + tx];
				sumI += pixelI;
				sumI2 += pixelI * pixelI;
				sumIT += pixelI * pixelT;
			}
		}

		double meanI = sumI / N;
		double stdDevI = sqrt(fmax(0.0, (sumI2 / N) - (meanI * meanI)));
		if (stdDevI < 1e-6) { // εҼ 񱳸    
			result[y * width + x] = -1.0f;
			return;
		}

		double ncc = (sumIT - N * meanI * meanT) * stdDevT_inv / stdDevI;
		result[y * width + x] = (float)ncc;
	}
 
	// ==========================================
	// --- FFT spectrum kernel 정의 ---
	// ==========================================
	__global__ void UcharToFloatKernel(const unsigned char* input, float* output, int N) {
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i < N) output[i] = (float)input[i];
	}
	__global__ void ComplexMultiplyKernel(hipfftComplex* a, const hipfftComplex* b, int N) {
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i < N) {
			float real_a = a[i].x;
			float imag_a = a[i].y;
			float real_b = b[i].x;
			float imag_b = b[i].y;
			a[i].x = real_a * real_b - imag_a * imag_b;
			a[i].y = real_a * imag_b + imag_a * real_b;
		}
	}
	__global__ void NormalizeAndConvertToUcharKernel(const float* input, unsigned char* output, int N) {
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i < N) {
			float val = input[i] / N; // IFFT  N  ȭ
			output[i] = (unsigned char)fmaxf(0.f, fminf(255.f, val));
		}
	}
	__global__ void FftShiftAndLogMagnitudeKernel(const hipfftComplex* fft_input, float* magnitude_output, int width, int height)
	{
		// 각 스레드가 담당할 출력 이미지의 좌표 (x, y)
		int x = blockIdx.x * blockDim.x + threadIdx.x;
		int y = blockIdx.y * blockDim.y + threadIdx.y;

		if (x >= width || y >= height) return;

		// FFT Shift 적용
		// 출력 좌표 (x, y)에 해당하는 주파수 좌표 (freq_x, freq_y)를 계산
		int freq_x = (x + width / 2) % width;
		int freq_y = (y + height / 2) % height;

		// R2C 포맷의 실제 데이터 너비
		int complex_width = width / 2 + 1;

		// 대칭성을 고려한 인덱스 계산 (수정된 핵심 로직)
		int lookup_x = freq_x;
		int lookup_y = freq_y;

		// 만약 계산하려는 주파수(freq_x)가 저장된 범위를 벗어난다면 (즉, 스펙트럼의 오른쪽 절반이라면),
		// 켤레 대칭 속성을 이용해 참조할 왼쪽의 좌표를 계산
		// |F(u, v)| = |F(-u, -v)|
		if (freq_x >= complex_width) {
			lookup_x = width - freq_x;
			lookup_y = (height - freq_y) % height; // y축도 함께 대칭 이동
		}

		// 최종적으로 R2C 데이터에서 값을 가져올 1차원 인덱스
		int input_idx = lookup_y * complex_width + lookup_x;

		// Magnitude 계산 및 Log 스케일 적용 
		hipfftComplex val = fft_input[input_idx];
		float magnitude = sqrtf(val.x * val.x + val.y * val.y);
		magnitude_output[y * width + x] = log10f(1.0f + magnitude);
	}
	// pixel is 0~255 validation
	__global__ void NormalizeFloatToUcharKernel(const float* float_input, unsigned char* uchar_output, int N, float min_val, float max_val)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if (i >= N) return;

		float range = max_val - min_val;
		if (range <= 1e-6f) {
			uchar_output[i] = 0;
			return;
		}
		float normalized_val = 255.0f * (float_input[i] - min_val) / range;
		uchar_output[i] = (unsigned char)fmaxf(0.f, fminf(255.f, normalized_val));
	}


	// ==========================================
	// --- Color-Contrast Launcher 함수 구현 ---
	// ==========================================
	bool LaunchBinarizationKernel(unsigned char* pixels, int width, int height, int stride, int threshold) {
		unsigned char* d_pixels = nullptr;
		size_t imageSize = (size_t)height * stride;
		CUDA_CHECK(hipMalloc(&d_pixels, imageSize));
		CUDA_CHECK(hipMemcpy(d_pixels, pixels, imageSize, hipMemcpyHostToDevice));
		dim3 block(16, 16);
		dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);
		BinarizationKernel << <grid, block >> > (d_pixels, width, height, stride, (unsigned char)threshold);
		CUDA_CHECK(hipGetLastError());
		CUDA_CHECK(hipDeviceSynchronize());
		CUDA_CHECK(hipMemcpy(pixels, d_pixels, imageSize, hipMemcpyDeviceToHost));
		hipFree(d_pixels);
		return true;
	}

	bool LaunchEqualizationKernel(unsigned char* pixels, int width, int height, int stride) {
		unsigned char* d_pixels = nullptr;
		unsigned int* d_hist = nullptr;
		int* d_cdf = nullptr;
		size_t imageSize = (size_t)height * stride;
		size_t histSize = 256 * sizeof(unsigned int);

		CUDA_CHECK(hipMalloc(&d_pixels, imageSize));
		CUDA_CHECK(hipMalloc(&d_hist, histSize));
		CUDA_CHECK(hipMemcpy(d_pixels, pixels, imageSize, hipMemcpyHostToDevice));
		CUDA_CHECK(hipMemset(d_hist, 0, histSize));

		dim3 block(16, 16);
		dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);
		Histogram256Kernel << <grid, block >> > (d_pixels, width, height, stride, d_hist);
		CUDA_CHECK(hipDeviceSynchronize());

		std::vector<unsigned int> h_hist(256);
		CUDA_CHECK(hipMemcpy(h_hist.data(), d_hist, histSize, hipMemcpyDeviceToHost));

		std::vector<int> h_cdf(256);
		h_cdf[0] = h_hist[0];
		for (int i = 1; i < 256; ++i) h_cdf[i] = h_cdf[i - 1] + h_hist[i];

		int cdfMin = 0;
		for (int i = 0; i < 256; ++i) {
			if (h_cdf[i] > 0) {
				cdfMin = h_cdf[i];
				break;
			}
		}

		CUDA_CHECK(hipMalloc(&d_cdf, 256 * sizeof(int)));
		CUDA_CHECK(hipMemcpy(d_cdf, h_cdf.data(), 256 * sizeof(int), hipMemcpyHostToDevice));

		EqualizationKernel << <grid, block >> > (d_pixels, width, height, stride, d_cdf, cdfMin, width * height);
		CUDA_CHECK(hipDeviceSynchronize());

		CUDA_CHECK(hipMemcpy(pixels, d_pixels, imageSize, hipMemcpyDeviceToHost));
		hipFree(d_pixels);
		hipFree(d_hist);
		hipFree(d_cdf);
		return true;
	}

	// ==========================================
	// --- Filter Launcher 함수 구현 ---
	// ==========================================
	bool LaunchGaussianBlurKernel(unsigned char* pixels, int width, int height, int stride, double sigma, int kernelSize, bool useCircularKernel) {
		if (kernelSize > 15) return false;
		unsigned char* d_input = nullptr, * d_output = nullptr;
		size_t imageSize = (size_t)height * stride;
		std::vector<float> h_kernel = createGaussianKernelFloat(kernelSize, sigma, useCircularKernel);

		CUDA_CHECK(hipMalloc(&d_input, imageSize));
		CUDA_CHECK(hipMalloc(&d_output, imageSize));
		CUDA_CHECK(hipMemcpy(d_input, pixels, imageSize, hipMemcpyHostToDevice));
		CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_filterKernel), h_kernel.data(), h_kernel.size() * sizeof(float)));

		dim3 block(TILE_DIM, TILE_DIM);
		dim3 grid((width + TILE_DIM - 1) / TILE_DIM, (height + TILE_DIM - 1) / TILE_DIM);
		ConvolutionSharedMemKernel << <grid, block >> > (d_input, d_output, width, height, stride, kernelSize);
		CUDA_CHECK(hipDeviceSynchronize());
		CUDA_CHECK(hipMemcpy(pixels, d_output, imageSize, hipMemcpyDeviceToHost));

		hipFree(d_input);
		hipFree(d_output);
		return true;
	}

	bool LaunchAverageBlurKernel(unsigned char* pixels, int width, int height, int stride, int kernelSize, bool useCircularKernel) {
		if (kernelSize > 15) return false;
		unsigned char* d_input = nullptr, * d_output = nullptr;
		size_t imageSize = (size_t)height * stride;
		std::vector<float> h_kernel = createAverageKernelFloat(kernelSize, useCircularKernel);

		CUDA_CHECK(hipMalloc(&d_input, imageSize));
		CUDA_CHECK(hipMalloc(&d_output, imageSize));
		CUDA_CHECK(hipMemcpy(d_input, pixels, imageSize, hipMemcpyHostToDevice));
		CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_filterKernel), h_kernel.data(), h_kernel.size() * sizeof(float)));

		dim3 block(TILE_DIM, TILE_DIM);
		dim3 grid((width + TILE_DIM - 1) / TILE_DIM, (height + TILE_DIM - 1) / TILE_DIM);
		ConvolutionSharedMemKernel << <grid, block >> > (d_input, d_output, width, height, stride, kernelSize);
		CUDA_CHECK(hipDeviceSynchronize());
		CUDA_CHECK(hipMemcpy(pixels, d_output, imageSize, hipMemcpyDeviceToHost));

		hipFree(d_input);
		hipFree(d_output);
		return true;
	}

	bool LaunchSobelKernel(unsigned char* pixels, int width, int height, int stride, int kernelSize) {
		if (kernelSize > 15) return false;
		unsigned char* d_input = nullptr, * d_output = nullptr;
		size_t imageSize = (size_t)height * stride;
		std::vector<float> h_kernelX, h_kernelY;
		createSobelKernelsFloat(h_kernelX, h_kernelY, kernelSize);

		CUDA_CHECK(hipMalloc(&d_input, imageSize));
		CUDA_CHECK(hipMalloc(&d_output, imageSize));
		CUDA_CHECK(hipMemcpy(d_input, pixels, imageSize, hipMemcpyHostToDevice));
		CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_sobelKernelX), h_kernelX.data(), h_kernelX.size() * sizeof(float)));
		CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_sobelKernelY), h_kernelY.data(), h_kernelY.size() * sizeof(float)));

		dim3 block(TILE_DIM, TILE_DIM);
		dim3 grid((width + TILE_DIM - 1) / TILE_DIM, (height + TILE_DIM - 1) / TILE_DIM);
		SobelSharedMemKernel << <grid, block >> > (d_input, d_output, width, height, stride, kernelSize);
		CUDA_CHECK(hipDeviceSynchronize());
		CUDA_CHECK(hipMemcpy(pixels, d_output, imageSize, hipMemcpyDeviceToHost));

		hipFree(d_input);
		hipFree(d_output);
		return true;
	}

	bool LaunchLaplacianKernel(unsigned char* pixels, int width, int height, int stride, int kernelSize) {
		if (kernelSize > 15) return false;
		unsigned char* d_input = nullptr, * d_output = nullptr;
		size_t imageSize = (size_t)height * stride;
		std::vector<float> h_kernel = createLaplacianKernelFloat(kernelSize);

		CUDA_CHECK(hipMalloc(&d_input, imageSize));
		CUDA_CHECK(hipMalloc(&d_output, imageSize));
		CUDA_CHECK(hipMemcpy(d_input, pixels, imageSize, hipMemcpyHostToDevice));
		CUDA_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_filterKernel), h_kernel.data(), h_kernel.size() * sizeof(float)));

		dim3 block(TILE_DIM, TILE_DIM);
		dim3 grid((width + TILE_DIM - 1) / TILE_DIM, (height + TILE_DIM - 1) / TILE_DIM);
		ConvolutionSharedMemKernel << <grid, block >> > (d_input, d_output, width, height, stride, kernelSize);
		CUDA_CHECK(hipDeviceSynchronize());
		CUDA_CHECK(hipMemcpy(pixels, d_output, imageSize, hipMemcpyDeviceToHost));

		hipFree(d_input);
		hipFree(d_output);
		return true;
	}

	// ==========================================
	// --- Morphology Launcher 함수 구현 ---
	// ==========================================
	bool LaunchDilationKernel(unsigned char* pixels, int width, int height, int stride, int kernelSize, bool useCircularKernel) {
		if (kernelSize > 15) return false;
		unsigned char* d_input = nullptr, * d_output = nullptr;
		size_t imageSize = (size_t)height * stride;

		CUDA_CHECK(hipMalloc(&d_input, imageSize));
		CUDA_CHECK(hipMalloc(&d_output, imageSize));
		CUDA_CHECK(hipMemcpy(d_input, pixels, imageSize, hipMemcpyHostToDevice));

		dim3 block(TILE_DIM, TILE_DIM);
		dim3 grid((width + TILE_DIM - 1) / TILE_DIM, (height + TILE_DIM - 1) / TILE_DIM);
		DilationSharedMemKernel << <grid, block >> > (d_input, d_output, width, height, stride, kernelSize, useCircularKernel);
		CUDA_CHECK(hipDeviceSynchronize());
		CUDA_CHECK(hipMemcpy(pixels, d_output, imageSize, hipMemcpyDeviceToHost));

		hipFree(d_input);
		hipFree(d_output);
		return true;
	}

	bool LaunchErosionKernel(unsigned char* pixels, int width, int height, int stride, int kernelSize, bool useCircularKernel) {
		if (kernelSize > 15) return false;
		unsigned char* d_input = nullptr, * d_output = nullptr;
		size_t imageSize = (size_t)height * stride;

		CUDA_CHECK(hipMalloc(&d_input, imageSize));
		CUDA_CHECK(hipMalloc(&d_output, imageSize));
		CUDA_CHECK(hipMemcpy(d_input, pixels, imageSize, hipMemcpyHostToDevice));

		dim3 block(TILE_DIM, TILE_DIM);
		dim3 grid((width + TILE_DIM - 1) / TILE_DIM, (height + TILE_DIM - 1) / TILE_DIM);
		ErosionSharedMemKernel << <grid, block >> > (d_input, d_output, width, height, stride, kernelSize, useCircularKernel);
		CUDA_CHECK(hipDeviceSynchronize());
		CUDA_CHECK(hipMemcpy(pixels, d_output, imageSize, hipMemcpyDeviceToHost));

		hipFree(d_input);
		hipFree(d_output);
		return true;
	}


	// ==========================================
   // --- NCC / SAD / SSD Launcher 함수 구현 ---
   // ==========================================

	bool LaunchSadKernel(const unsigned char* image, int width, int height, int stride,
		const unsigned char* templ, int tempWidth, int tempHeight, int tempStride,
		int* out_x, int* out_y)
	{
		// GPU 메모리 할당 및 데이터 복사
		unsigned char* d_image = nullptr, * d_templ = nullptr;
		unsigned int* d_result = nullptr;
		CUDA_CHECK(hipMalloc(&d_image, (size_t)height * stride));
		CUDA_CHECK(hipMalloc(&d_templ, (size_t)tempHeight * tempStride));
		CUDA_CHECK(hipMalloc(&d_result, (size_t)width * height * sizeof(unsigned int)));
		CUDA_CHECK(hipMemcpy(d_image, image, (size_t)height * stride, hipMemcpyHostToDevice));
		CUDA_CHECK(hipMemcpy(d_templ, templ, (size_t)tempHeight * tempStride, hipMemcpyHostToDevice));

		// SAD 커널 실행
		dim3 block(16, 16);
		dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);
		SadKernel << <grid, block >> > (d_image, d_templ, d_result, width, height, stride, tempWidth, tempHeight, tempStride);
		CUDA_CHECK(hipDeviceSynchronize());

		// 결과를 CPU로 복사하여 최소값 찾기
		std::vector<unsigned int> h_result(width * height);
		CUDA_CHECK(hipMemcpy(h_result.data(), d_result, (size_t)width * height * sizeof(unsigned int), hipMemcpyDeviceToHost));

		unsigned int minSad = -1; // unsigned int의 최대값으로 초기화
		*out_x = 0; *out_y = 0;
		for (int y = 0; y <= height - tempHeight; ++y) {
			for (int x = 0; x <= width - tempWidth; ++x) {
				if (h_result[y * width + x] < minSad) {
					minSad = h_result[y * width + x];
					*out_x = x;
					*out_y = y;
				}
			}
		}

		// 메모리 해제
		hipFree(d_image);
		hipFree(d_templ);
		hipFree(d_result);
		return true;
	}

	bool LaunchSsdKernel(const unsigned char* image, int width, int height, int stride,
		const unsigned char* templ, int tempWidth, int tempHeight, int tempStride,
		int* out_x, int* out_y)
	{
		// GPU 메모리 할당 및 데이터 복사
		unsigned char* d_image = nullptr, * d_templ = nullptr;
		unsigned int* d_result = nullptr;
		CUDA_CHECK(hipMalloc(&d_image, (size_t)height * stride));
		CUDA_CHECK(hipMalloc(&d_templ, (size_t)tempHeight * tempStride));
		CUDA_CHECK(hipMalloc(&d_result, (size_t)width * height * sizeof(unsigned int)));
		CUDA_CHECK(hipMemcpy(d_image, image, (size_t)height * stride, hipMemcpyHostToDevice));
		CUDA_CHECK(hipMemcpy(d_templ, templ, (size_t)tempHeight * tempStride, hipMemcpyHostToDevice));

		// SSD 커널 실행
		dim3 block(16, 16);
		dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);
		SsdKernel << <grid, block >> > (d_image, d_templ, d_result, width, height, stride, tempWidth, tempHeight, tempStride);
		CUDA_CHECK(hipDeviceSynchronize());

		// 결과를 CPU로 복사하여 최소값 찾기
		std::vector<unsigned int> h_result(width * height);
		CUDA_CHECK(hipMemcpy(h_result.data(), d_result, (size_t)width * height * sizeof(unsigned int), hipMemcpyDeviceToHost));

		unsigned int minSsd = -1; // unsigned int의 최대값으로 초기화
		*out_x = 0; *out_y = 0;
		for (int y = 0; y <= height - tempHeight; ++y) {
			for (int x = 0; x <= width - tempWidth; ++x) {
				if (h_result[y * width + x] < minSsd) {
					minSsd = h_result[y * width + x];
					*out_x = x;
					*out_y = y;
				}
			}
		}

		// 메모리 해제
		hipFree(d_image);
		hipFree(d_templ);
		hipFree(d_result);
		return true;
	}

	bool LaunchNccKernel(const unsigned char* image, int width, int height, int stride,
		const unsigned char* templ, int tempWidth, int tempHeight, int tempStride,
		int* out_x, int* out_y) {
		// 1. CPU ø հ ǥ 
		double sumT = 0.0, sumT2 = 0.0;
		int N = tempWidth * tempHeight;
		for (int y = 0; y < tempHeight; ++y) {
			for (int x = 0; x < tempWidth; ++x) {
				sumT += templ[y * tempStride + x];
				sumT2 += templ[y * tempStride + x] * templ[y * tempStride + x];
			}
		}
		double meanT = sumT / N;
		double stdDevT = sqrt(fmax(0.0, (sumT2 / N) - (meanT * meanT)));
		if (stdDevT < 1e-6) return false;

		// GPU 
		unsigned char* d_image = nullptr, * d_templ = nullptr;
		float* d_result = nullptr;
		CUDA_CHECK(hipMalloc(&d_image, (size_t)height * stride));
		CUDA_CHECK(hipMalloc(&d_templ, (size_t)tempHeight * tempStride));
		CUDA_CHECK(hipMalloc(&d_result, (size_t)width * height * sizeof(float)));
		CUDA_CHECK(hipMemcpy(d_image, image, (size_t)height * stride, hipMemcpyHostToDevice));
		CUDA_CHECK(hipMemcpy(d_templ, templ, (size_t)tempHeight * tempStride, hipMemcpyHostToDevice));

		// NCC 
		dim3 block(16, 16);
		dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);
		NccKernel << <grid, block >> > (d_image, d_templ, d_result, width, height, stride, tempWidth, tempHeight, tempStride, meanT, 1.0 / stdDevT);
		CUDA_CHECK(hipDeviceSynchronize());

		// CPU 
		std::vector<float> h_result(width * height);
		CUDA_CHECK(hipMemcpy(h_result.data(), d_result, (size_t)width * height * sizeof(float), hipMemcpyDeviceToHost));

		float maxNcc = -2.0f;
		*out_x = 0; *out_y = 0;
		for (int y = 0; y <= height - tempHeight; ++y) {
			for (int x = 0; x <= width - tempWidth; ++x) {
				if (h_result[y * width + x] > maxNcc) {
					maxNcc = h_result[y * width + x];
					*out_x = x;
					*out_y = y;
				}
			}
		}

		// Memfree
		hipFree(d_image);
		hipFree(d_templ);
		hipFree(d_result);
		return true;
	}




	bool LaunchFftFilterKernel(unsigned char* pixels, int width, int height, int stride, int kernelSize) {
		if (width != stride) return false;

		const int N = width * height;
		const int complexWidth = (width / 2 + 1);
		const std::vector<float>& filterKernel = createGaussianKernelFloat(kernelSize, 2, false);
		hipfftHandle planR2C, planC2R;
		float* d_input_float = nullptr, * d_kernel_float = nullptr;
		hipfftComplex* d_input_complex = nullptr, * d_kernel_complex = nullptr;

		// --- MODIFIED: cufft... Լ CUFFT_CHECK  ---
		CUFFT_CHECK(hipfftPlan2d(&planR2C, height, width, HIPFFT_R2C));
		CUFFT_CHECK(hipfftPlan2d(&planC2R, height, width, HIPFFT_C2R));

		std::vector<float> h_padded_kernel(N, 0.0f);
		int center = kernelSize / 2;
		for (int y = 0; y < kernelSize; ++y) {
			for (int x = 0; x < kernelSize; ++x) {
				int shifted_y = (y - center + height) % height;
				int shifted_x = (x - center + width) % width;
				h_padded_kernel[shifted_y * width + shifted_x] = filterKernel[y * kernelSize + x];
			}
		}

		CUDA_CHECK(hipMalloc(&d_input_float, N * sizeof(float)));
		CUDA_CHECK(hipMalloc(&d_kernel_float, N * sizeof(float)));
		CUDA_CHECK(hipMalloc(&d_input_complex, complexWidth * height * sizeof(hipfftComplex)));
		CUDA_CHECK(hipMalloc(&d_kernel_complex, complexWidth * height * sizeof(hipfftComplex)));

		dim3 block(256);
		dim3 grid((N + block.x - 1) / block.x);
		UcharToFloatKernel << <grid, block >> > (pixels, d_input_float, N);

		CUDA_CHECK(hipMemcpy(d_kernel_float, h_padded_kernel.data(), N * sizeof(float), hipMemcpyHostToDevice));

		// --- MODIFIED: cufft... Լ CUFFT_CHECK  ---
		CUFFT_CHECK(hipfftExecR2C(planR2C, d_input_float, d_input_complex));
		CUFFT_CHECK(hipfftExecR2C(planR2C, d_kernel_float, d_kernel_complex));

		dim3 complex_grid((complexWidth * height + block.x - 1) / block.x);
		ComplexMultiplyKernel << <complex_grid, block >> > (d_input_complex, d_kernel_complex, complexWidth * height);

		// --- MODIFIED: cufft... Լ CUFFT_CHECK  ---
		CUFFT_CHECK(hipfftExecC2R(planC2R, d_input_complex, d_input_float));

		NormalizeAndConvertToUcharKernel << <grid, block >> > (d_input_float, pixels, N);
		CUDA_CHECK(hipDeviceSynchronize());

		// --- MODIFIED: cufft... Լ CUFFT_CHECK  ---
		CUFFT_CHECK(hipfftDestroy(planR2C));
		CUFFT_CHECK(hipfftDestroy(planC2R));
		hipFree(d_input_float);
		hipFree(d_kernel_float);
		hipFree(d_input_complex);
		hipFree(d_kernel_complex);

		return true;
	}

	bool LaunchFftSpectrumKernel(unsigned char* pixels, int width, int height, int stride) {
		if (width != stride) return false;

		const int N = width * height;
		const int complexWidth = (width / 2 + 1);

		hipfftHandle planR2C;
		float* d_input_float = nullptr;
		float* d_magnitude_float = nullptr;
		hipfftComplex* d_input_complex = nullptr;
		CUFFT_CHECK(hipfftPlan2d(&planR2C, height, width, HIPFFT_R2C));
		CUDA_CHECK(hipMalloc(&d_input_float, N * sizeof(float)));
		CUDA_CHECK(hipMalloc(&d_magnitude_float, N * sizeof(float)));
		CUDA_CHECK(hipMalloc(&d_input_complex, complexWidth * height * sizeof(hipfftComplex)));

		dim3 grid((width + 15) / 16, (height + 15) / 16);
		dim3 block(16, 16);

		// GPU float 
		dim3 grid1D((N + 255) / 256);
		dim3 block1D(256);
		UcharToFloatKernel << <grid1D, block1D >> > (pixels, d_input_float, N);

		// Apply
		CUFFT_CHECK(hipfftExecR2C(planR2C, d_input_float, d_input_complex));

		// Shifting - Apply Log to magnitude
		FftShiftAndLogMagnitudeKernel << <grid, block >> > (d_input_complex, d_magnitude_float, width, height);

		//CPU Min/Max 
		std::vector<float> h_magnitude(N);
		CUDA_CHECK(hipMemcpy(h_magnitude.data(), d_magnitude_float, N * sizeof(float), hipMemcpyDeviceToHost));

		// std::minmax_element 
		auto result_pair = std::minmax_element(h_magnitude.begin(), h_magnitude.end());
		auto min_it = result_pair.first;
		auto max_it = result_pair.second;

		float min_val = *min_it;
		float max_val = *max_it;

		// Magnitude to Pixel  
		NormalizeFloatToUcharKernel << <grid1D, block1D >> > (d_magnitude_float, pixels, N, min_val, max_val);
		CUDA_CHECK(hipDeviceSynchronize());

		// 
		CUFFT_CHECK(hipfftDestroy(planR2C));
		hipFree(d_input_float);
		hipFree(d_magnitude_float);
		hipFree(d_input_complex);

		return true;
	}



}
